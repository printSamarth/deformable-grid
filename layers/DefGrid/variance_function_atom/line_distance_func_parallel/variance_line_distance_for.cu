#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <THC/THC.h>
#include <vector>
#include <torch/torch.h>
#include <torch/extension.h>

#define eps 1e-10
#define SCALE 1.0
#define MAX_DIS  9999999999.0
#include <sys/time.h>

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

//extern THCState * state;

template<typename scalar_t>
__host__ __device__ scalar_t line_variance_parallel_cuda_abs(scalar_t a){
	if (a > 0.0){
		return a;
	}
	else{
		return -a;
	}
}

template<typename scalar_t>
__host__ __device__ scalar_t line_variance_parallel_cuda_square(scalar_t a){
	return a * a;
}

template<typename scalar_t>
__host__ __device__ scalar_t line_variance_parallel_cuda_divide_non_zero(scalar_t a){
	if (a == 0){
		return eps;
	}
	if (a < 0){
		return a - eps;
	}
	if (a > 0){
		return a + eps;
	}
	return eps;
}

template<typename scalar_t>
__host__ __device__ scalar_t line_variance_parallel_cuda_min_dis(scalar_t a, scalar_t b, scalar_t c){
	scalar_t min_d = a;
	if (b < min_d){
		min_d = b;
	}
	if (c < min_d){
		min_d = c;
	}
	return min_d;
}

template<typename scalar_t>
__host__ __device__ scalar_t line_variance_parallel_cuda_min_dis_idx(scalar_t a, scalar_t b, scalar_t c){
	scalar_t min_d = a;
	int min_idx = 0;
	if (b < min_d){
		min_d = b;
		min_idx = 1;
	}
	if (c < min_d){
		min_d = c;
		min_idx = 2;
	}
	return min_idx;
}

template <typename scalar_t>
__host__ __device__ scalar_t distance_line(scalar_t x1, scalar_t y1, scalar_t x2, scalar_t y2, scalar_t x, scalar_t y){
	
	scalar_t dx1x2 = -x1 + x2;
	scalar_t dy1y2 = -y1 + y2;
	scalar_t dx1x = x - x1;
	scalar_t dy1y = y - y1;
	
	scalar_t c1 = - x * x1 + x * x2 + x1 * x1 - x1 * x2 - y * y1 + y * y2 + y1 * y1 - y1 * y2;
	scalar_t c2 = x1 * x1 - 2 * x1 * x2 + x2 * x2 + y1 * y1  - 2 * y1 * y2 + y2 * y2;
	
	scalar_t d1 = -dx1x + dx1x2 * c1 / line_variance_parallel_cuda_divide_non_zero(c2);
	scalar_t d2 = -dy1y + dy1y2 * c1 / line_variance_parallel_cuda_divide_non_zero(c2);
	
	scalar_t dis = 	line_variance_parallel_cuda_abs(d1)	+ line_variance_parallel_cuda_abs(d2);

	return dis;
}
template <typename scalar_t>
__host__ __device__ scalar_t distance_point(scalar_t x1, scalar_t y1, scalar_t x, scalar_t y){
	return line_variance_parallel_cuda_abs(x - x1) + line_variance_parallel_cuda_abs(y - y1);
}

template <typename scalar_t>
__host__ __device__ void distance(scalar_t* ret, scalar_t x1, scalar_t y1, scalar_t x2, scalar_t y2, scalar_t x3, scalar_t y3, scalar_t x, scalar_t y)
{
	//https://en.wikipedia.org/wiki/Barycentric_coordinate_system
	
	scalar_t x1_x2 = x1 - x2;
	scalar_t y1_y2 = y1 - y2;
	scalar_t x1_x3 = x1 - x3;
	scalar_t y1_y3 = y1 - y3;
	scalar_t x2_x3 = x2 - x3;
	scalar_t y2_y3 = y2 - y3;
	
	scalar_t x_x1 = x - x1;
	scalar_t y_y1 = y - y1;
	scalar_t x_x2 = x - x2;
	scalar_t y_y2 = y - y2;
	scalar_t x_x3 = x - x3;
	scalar_t y_y3 = y - y3;

	scalar_t k1 = y2_y3 * x_x3 - x2_x3 * y_y3;
	scalar_t k2 = x1_x3 * y_y3 - y1_y3 * x_x3;
	scalar_t k3 = y2_y3 * x1_x3 - x2_x3 * y1_y3;
	
	if(k3 == 0){ // not a legal triangle
		ret[0] = -2;
		return;
	}
	if(k3 > 0){ // clock-wise triangle
		ret[0] = -1;
		return;
	}

	//scalar_t l1 = k1 / line_variance_parallel_cuda_divide_non_zero(k3);
	//scalar_t l2 = k2 / line_variance_parallel_cuda_divide_non_zero(k3);
	scalar_t l1 = k1 / k3;
	scalar_t l2 = k2 / k3;
	scalar_t l3 = 1 - l1 - l2;

	scalar_t dis12 = distance_line(x1, y1, x2, y2, x, y);
	scalar_t dis23 = distance_line(x2, y2, x3, y3, x, y);
	scalar_t dis13 = distance_line(x1, y1, x3, y3, x, y);
	
	if (l1 >= 0 && l2 >= 0 && l3 >= 0){ // lie inside or on the boundary
		scalar_t min_dis_line = line_variance_parallel_cuda_min_dis(dis12, dis23, dis13);
		scalar_t min_dis_line_idx = line_variance_parallel_cuda_min_dis_idx(dis12, dis23, dis13);
		ret[0] = 0;
		ret[1] = min_dis_line;
		ret[2] = min_dis_line_idx;
		return;
	}

	// whether point can calculate distance to certain line
	bool within12 = ((y1_y2 * y_y1 + x_x1 * x1_x2) * (y1_y2 * y_y2 + x_x2 * x1_x2)) <= 0;
	bool within23 = ((y2_y3 * y_y3 + x_x3 * x2_x3) * (y2_y3 * y_y2 + x_x2 * x2_x3)) <= 0;	
	bool within13 = ((y1_y3 * y_y1 + x_x1 * x1_x3) * (y1_y3 * y_y3 + x_x3 * x1_x3)) <= 0;

	dis12 = within12 ? dis12 : MAX_DIS;
	dis23 = within23 ? dis23 : MAX_DIS;
	dis13 = within13 ? dis13 : MAX_DIS;

	scalar_t min_dis_line = line_variance_parallel_cuda_min_dis(dis12, dis23, dis13);
	scalar_t min_dis_line_idx = line_variance_parallel_cuda_min_dis_idx(dis12, dis23, dis13);
	
	scalar_t d1 = distance_point(x1, y1, x, y);
	scalar_t d2 = distance_point(x2, y2, x, y);  
	scalar_t d3 = distance_point(x3, y3, x, y);
	
	scalar_t min_dis_point = line_variance_parallel_cuda_min_dis(d1, d2, d3);
	scalar_t min_dis_point_idx = line_variance_parallel_cuda_min_dis_idx(d1, d2, d3);

	if (min_dis_line < min_dis_point){
		ret[0] = 1;
		ret[1] = min_dis_line;
		ret[2] = min_dis_line_idx;
	}
	else{
		ret[0] = 2;
		ret[1] = min_dis_point;
		ret[2] = min_dis_point_idx;
	}

}

template <typename scalar_t> 
__global__ void line_variance_parallel_cuda_forward_kernel_batch(
        const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> img_fea_bxnxd,
        const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> grid_fea_bxkxd,
		const torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> grid_bxkx3x2,
		const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> img_pos_bxnx2,
		torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> variance_bxn,
		torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> reconstruct_bxnxd,
		torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> buffer_bxnxk,
		int bnum, int n_pixel, int n_grid, int d_fea, float sigma
)
{
	// bidx * height + heiidx
	int presentthread = blockIdx.x * blockDim.x + threadIdx.x;
	int pixel_idx = presentthread % n_pixel;
	int bidx = (presentthread - pixel_idx) / n_pixel;


	if (bidx >= bnum || pixel_idx >= n_pixel) {
		return;
	}

	/////////////////////////////////////////////////////////////////
	// which pixel it belongs to
	int total_idx = bidx * n_pixel * n_grid + pixel_idx * n_grid;

	scalar_t pixel_x = img_pos_bxnx2[bidx][pixel_idx][0];
	scalar_t pixel_y = img_pos_bxnx2[bidx][pixel_idx][1];

	scalar_t x0 = pixel_x * SCALE;
	scalar_t y0 = pixel_y * SCALE;
	scalar_t min_distance = 0.0;
	scalar_t find_sign = 0.0;
	scalar_t sum_exp = 0.0;
    scalar_t max_dist = -MAX_DIS;
    scalar_t ax, ay, bx, by, cx, cy;
	scalar_t condition;
	int img_pos_total_idx = bidx * n_pixel * 2 + pixel_idx * 2;
	scalar_t ret[3] = {0};
	for (int grididx = 0; grididx < n_grid; grididx++){

		ax = grid_bxkx3x2[bidx][grididx][0][0] * SCALE;
		ay = grid_bxkx3x2[bidx][grididx][0][1] * SCALE;
		bx = grid_bxkx3x2[bidx][grididx][1][0] * SCALE;
		by = grid_bxkx3x2[bidx][grididx][1][1] * SCALE;
		cx = grid_bxkx3x2[bidx][grididx][2][0] * SCALE;
		cy = grid_bxkx3x2[bidx][grididx][2][1] * SCALE;

		distance(ret, ax, ay, bx, by, cx, cy, x0, y0);
		condition = ret[0];
		min_distance = ret[1];

		if (condition < 0) {
			min_distance = - MAX_DIS;
		}
		else if (condition == 0 && find_sign == 0){
			min_distance = min_distance / sigma;
			find_sign == 1;
		}
		else{
			min_distance = - min_distance / sigma;
		}

		max_dist = max_dist > min_distance ? max_dist : min_distance;
		buffer_bxnxk[bidx][pixel_idx][grididx] = min_distance;
	}

	for (int grididx = 0; grididx < n_grid; grididx++){
	    buffer_bxnxk[bidx][pixel_idx][grididx] = expf(buffer_bxnxk[bidx][pixel_idx][grididx] - max_dist);
		sum_exp += buffer_bxnxk[bidx][pixel_idx][grididx];
	}

	scalar_t variance = 0.0;
	scalar_t grid_f = 0.0;
	scalar_t pixel_f = 0.0;
	scalar_t diff = 0.0;
	scalar_t w = 0.0;
	scalar_t difference = 0.0;
	for (int grididx = 0; grididx < n_grid; grididx++){
		int in_sign = 0;
		if(buffer_bxnxk[bidx][pixel_idx][grididx] == 1){
			in_sign = 1;
		}
        buffer_bxnxk[bidx][pixel_idx][grididx] = buffer_bxnxk[bidx][pixel_idx][grididx] / (sum_exp + 1e-15);
		w = buffer_bxnxk[bidx][pixel_idx][grididx];
		difference = 0.0;
		for (int d = 0; d < d_fea; d++){
			grid_f = grid_fea_bxkxd[bidx][grididx][d];
			pixel_f = img_fea_bxnxd[bidx][pixel_idx][d];
			reconstruct_bxnxd[bidx][pixel_idx][d] += w * grid_f;
			diff = line_variance_parallel_cuda_square(grid_f - pixel_f);
			difference = difference + diff;
		}
		variance = variance + w * difference;
		if(in_sign == 1){ //hard variance for upsample
			buffer_bxnxk[bidx][pixel_idx][grididx] = difference;
		}
		else{
			buffer_bxnxk[bidx][pixel_idx][grididx] = 0;
		}
	}
	variance_bxn[bidx][pixel_idx] = variance;
}

template <typename scalar_t>
__global__ void line_variance_parallel_cuda_forward_kernel_batch_calc_buffer(
        const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> img_fea_bxnxd,
        const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> grid_fea_bxkxd,
		const torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> grid_bxkx3x2,
		const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> img_pos_bxnx2,
		torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> variance_bxn,
		torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> reconstruct_bxnxd,
		torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> buffer_bxnxk,
		int bnum, int n_pixel, int n_grid, int d_fea, float sigma
)
{
	// bidx * height + heiidx
	int presentthread = blockIdx.x * blockDim.x + threadIdx.x;
	int grididx = presentthread % n_grid;
	int pixel_idx = (presentthread - grididx) / n_grid;
	int bidx = 0;


	if (bidx >= bnum || pixel_idx >= n_pixel || grididx >= n_grid) {
		return;
	}

	/////////////////////////////////////////////////////////////////
	// which pixel it belongs to


	scalar_t pixel_x = img_pos_bxnx2[bidx][pixel_idx][0];
	scalar_t pixel_y = img_pos_bxnx2[bidx][pixel_idx][1];

	scalar_t x0 = pixel_x * SCALE;
	scalar_t y0 = pixel_y * SCALE;
	scalar_t min_distance = 0.0;
    scalar_t ax, ay, bx, by, cx, cy;
	scalar_t condition;
	scalar_t ret[3] = {0};

    ax = grid_bxkx3x2[bidx][grididx][0][0] * SCALE;
    ay = grid_bxkx3x2[bidx][grididx][0][1] * SCALE;
    bx = grid_bxkx3x2[bidx][grididx][1][0] * SCALE;
    by = grid_bxkx3x2[bidx][grididx][1][1] * SCALE;
    cx = grid_bxkx3x2[bidx][grididx][2][0] * SCALE;
    cy = grid_bxkx3x2[bidx][grididx][2][1] * SCALE;

    distance(ret, ax, ay, bx, by, cx, cy, x0, y0);
    condition = ret[0];
    min_distance = ret[1];

    if (condition < 0) {
        min_distance = - MAX_DIS;
    }
    else if (condition == 0){
        min_distance = min_distance / sigma;
    }
    else{
        min_distance = - min_distance / sigma;
    }
    buffer_bxnxk[bidx][pixel_idx][grididx] = min_distance;
}

template <typename scalar_t>
__global__ void line_variance_parallel_cuda_forward_kernel_batch_max_sum(
        const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> img_fea_bxnxd,
        const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> grid_fea_bxkxd,
		const torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> grid_bxkx3x2,
		const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> img_pos_bxnx2,
		torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> variance_bxn,
		torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> reconstruct_bxnxd,
		torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> buffer_bxnxk,
		scalar_t* __restrict__ buffer_bxn,
		int bnum, int n_pixel, int n_grid, int d_fea, float sigma
)
{
	// bidx * height + heiidx
	int presentthread = blockIdx.x * blockDim.x + threadIdx.x;
	int pixel_idx = presentthread % n_pixel;
	int bidx = (presentthread - pixel_idx) / n_pixel;

	if (bidx >= bnum || pixel_idx >= n_pixel) {
		return;
	}
	scalar_t min_distance = 0.0;
	scalar_t find_sign = 0.0;
	scalar_t sum_exp = 0.0;
    scalar_t max_dist = -MAX_DIS;

    for (int grididx = 0; grididx < n_grid; grididx++){
        min_distance = buffer_bxnxk[bidx][pixel_idx][grididx];
//        if (find_sign == 1 && min_distance > 0){
//            min_distance = -min_distance;
//        }
//        if (min_distance > 0){
//            find_sign = 1;
//        }
        max_dist = max_dist > min_distance ? max_dist : min_distance;
		buffer_bxnxk[bidx][pixel_idx][grididx] = min_distance;
    }

	for (int grididx = 0; grididx < n_grid; grididx++){
	    buffer_bxnxk[bidx][pixel_idx][grididx] = expf(buffer_bxnxk[bidx][pixel_idx][grididx] - max_dist);
		sum_exp += buffer_bxnxk[bidx][pixel_idx][grididx];
	}
	buffer_bxn[bidx * n_pixel + pixel_idx] = sum_exp;
}


#define BLOCK_SIZE 1024
#define WARP_SIZE 32

template <typename scalar_t>
__inline__ __device__ scalar_t warpReduceSum(scalar_t val) {
    for (int offset = WARP_SIZE/2; offset > 0; offset /= 2){
        val += __shfl_down(val, offset);
    }
    return val;
}

template <typename scalar_t>
__global__ void blockReduceSum(
        scalar_t* __restrict__ buffer_bxnxk,
		scalar_t* __restrict__ buffer_bxnx4,
		scalar_t* __restrict__ max_dist_bxn,
		int bnum, int n_pixel, int n_grid, int split_size) {
    static __shared__ scalar_t shared[BLOCK_SIZE / WARP_SIZE]; // Shared mem for 32 partial sums
    int lane = threadIdx.x % WARP_SIZE;
    int wid = threadIdx.x / WARP_SIZE;

    // each thread loads one element from global to local register
    int presentthread = blockIdx.x*blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int pixel_idx = (presentthread - tid) / (BLOCK_SIZE * split_size);
    int block_idx = blockIdx.x;
    int split = block_idx % split_size;
    int grididx = split * BLOCK_SIZE + tid;

	int bidx = 0;
    if (tid == 0 && pixel_idx < n_pixel) buffer_bxnx4[bidx * n_pixel * split_size + pixel_idx * split_size + split] = 0;
    scalar_t val = 0.0;

    if (bidx < bnum && pixel_idx < n_pixel && grididx < n_grid){
        scalar_t max_dist = max_dist_bxn[bidx * n_pixel + pixel_idx];
        buffer_bxnxk[bidx * n_pixel * n_grid + pixel_idx * n_grid + grididx] = expf(buffer_bxnxk[bidx * n_pixel * n_grid + pixel_idx * n_grid + grididx] - max_dist);
        val = buffer_bxnxk[bidx * n_pixel * n_grid + pixel_idx * n_grid + grididx];
    }
    val = warpReduceSum(val);     // Each warp performs partial reduction
    if (lane==0) shared[wid]=val; // Write reduced value to shared memory
     __syncthreads();              // Wait for all partial reductions
    //read from shared memory only if that warp existed
    val = (threadIdx.x < blockDim.x / WARP_SIZE) ? shared[lane] : 0;
    if (wid==0){
        val = warpReduceSum(val); //Final reduce within first warp
        if (tid == 0 && pixel_idx < n_pixel){
            buffer_bxnx4[bidx * n_pixel * split_size + pixel_idx * split_size + split] = val;
        }
    }
}

template <typename scalar_t>
__inline__ __device__ scalar_t warpReduceMax(scalar_t val) {
    for (int offset = WARP_SIZE/2; offset > 0; offset /= 2){
        val = max(val, __shfl_down(val, offset));
    }
    return val;
}

template <typename scalar_t>
__global__ void blockReduceMax(
        scalar_t* __restrict__ buffer_bxnxk,
		scalar_t* __restrict__ buffer_bxnx4,
		int bnum, int n_pixel, int n_grid, int split_size)
		{
    static __shared__ scalar_t shared[BLOCK_SIZE / WARP_SIZE]; // Shared mem for 32 partial sums
    int lane = threadIdx.x % WARP_SIZE;
    int wid = threadIdx.x / WARP_SIZE;
    // each thread loads one element from global to local register
    int presentthread = blockIdx.x*blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int pixel_idx = (presentthread - tid) / (BLOCK_SIZE * split_size);
    int block_idx = blockIdx.x;
    int split = block_idx % split_size;
    int grididx = split * BLOCK_SIZE + tid;

	int bidx = 0;
    if (tid == 0 && pixel_idx < n_pixel) buffer_bxnx4[bidx * n_pixel * split_size + pixel_idx * split_size + split] = -MAX_DIS;
    scalar_t val = -MAX_DIS;

    if (bidx < bnum && pixel_idx < n_pixel && grididx < n_grid){
        val = buffer_bxnxk[bidx * n_pixel * n_grid + pixel_idx * n_grid + grididx];
    }

    val = warpReduceMax(val);     // Each warp performs partial reduction
    if (lane==0) shared[wid]=val; // Write reduced value to shared memory
     __syncthreads();              // Wait for all partial reductions
    //read from shared memory only if that warp existed
    val = (threadIdx.x < blockDim.x / WARP_SIZE) ? shared[lane] : -MAX_DIS;
    if (wid==0){
        val = warpReduceMax(val); //Final reduce within first warp
        if (tid == 0 && pixel_idx < n_pixel){
            buffer_bxnx4[bidx * n_pixel * split_size + pixel_idx * split_size + split] = val;
        }
    }
}

template <typename scalar_t>
__global__ void line_variance_parallel_cuda_forward_kernel_batch_max_reduce_last_step(
		scalar_t* __restrict__ buffer_bxnx4,
		scalar_t* __restrict__ buffer_bxn,
		int bnum, int n_pixel, int n_grid, int split_size)
{
   // bidx * height + heiidx
	int presentthread = blockIdx.x * blockDim.x + threadIdx.x;
	int pixel_idx = presentthread % n_pixel;
	int bidx = (presentthread - pixel_idx) / n_pixel;

	if (bidx >= bnum || pixel_idx >= n_pixel) {
		return;
	}
    int base_idx = bidx * n_pixel * split_size + pixel_idx * split_size;
    scalar_t max_v = buffer_bxnx4[base_idx + 0];
    for (int t=1; t < split_size; t++){
        if(buffer_bxnx4[base_idx + t] > max_v){
            max_v = buffer_bxnx4[base_idx + t];
        }
    }
    buffer_bxn[bidx * n_pixel + pixel_idx] = max_v;
}

template <typename scalar_t>
__global__ void line_variance_parallel_cuda_forward_kernel_batch_sum_reduce_last_step(
		scalar_t* __restrict__ buffer_bxnx4,
		scalar_t* __restrict__ buffer_bxn,
		int bnum, int n_pixel, int n_grid, int split_size)
{
   // bidx * height + heiidx
	int presentthread = blockIdx.x * blockDim.x + threadIdx.x;
	int pixel_idx = presentthread % n_pixel;
	int bidx = (presentthread - pixel_idx) / n_pixel;

	if (bidx >= bnum || pixel_idx >= n_pixel) {
		return;
	}
    int base_idx = bidx * n_pixel * split_size + pixel_idx * split_size;
    scalar_t sum_v = buffer_bxnx4[base_idx + 0];
    for (int t=1; t < split_size; t++){
        sum_v += buffer_bxnx4[base_idx + t];
    }
    buffer_bxn[bidx * n_pixel + pixel_idx] = sum_v;
}

template <typename scalar_t>
__global__ void line_variance_parallel_cuda_forward_kernel_batch_final_1(
        const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> img_fea_bxnxd,
        const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> grid_fea_bxkxd,
		const torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> grid_bxkx3x2,
		const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> img_pos_bxnx2,
		torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> variance_bxn,
		torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> buffer_bxnxk,
		scalar_t* __restrict__ buffer_bxn,
		scalar_t* __restrict__ reconstruct_buffer_bxnxdx4,
		int bnum, int n_pixel, int n_grid, int d_fea, int fea_idx, int split_size)
{
    static __shared__ scalar_t shared[BLOCK_SIZE / WARP_SIZE]; // Shared mem for 32 partial sums
    int lane = threadIdx.x % WARP_SIZE;
    int wid = threadIdx.x / WARP_SIZE;

    int presentthread = blockIdx.x*blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int pixel_idx = (presentthread - tid) / (BLOCK_SIZE * split_size); // N_allthread = n_pixel * 1 * BLOCK_SIZE * split_size
    int block_idx = blockIdx.x;
    int split_idx = block_idx % split_size;
    int grididx = split_idx * BLOCK_SIZE + tid;
	int bidx = 0;

	scalar_t grid_f = 0.0;
    scalar_t pixel_f = 0.0;
    scalar_t diff = 0.0;
    scalar_t w = 0.0;
    scalar_t difference = 0.0;
    scalar_t sum_exp = 0.0;

	// We first calculate the difference for the buffer:
	if (grididx < n_grid && fea_idx == 0 && pixel_idx < n_pixel){ // first run calculate the buffer (soft assignment
        sum_exp = buffer_bxn[bidx * n_pixel + pixel_idx];
        buffer_bxnxk[bidx][pixel_idx][grididx] = buffer_bxnxk[bidx][pixel_idx][grididx] / (sum_exp + 1e-15);
	}

	// shared add for reconstruct
    if (tid == 0 && pixel_idx < n_pixel && fea_idx < d_fea) reconstruct_buffer_bxnxdx4[pixel_idx * d_fea * split_size + fea_idx * split_size + split_idx] = 0;
    scalar_t val = 0.0;
    if (pixel_idx < n_pixel && grididx < n_grid){
        w = buffer_bxnxk[bidx][pixel_idx][grididx];
        grid_f = grid_fea_bxkxd[bidx][grididx][fea_idx];
        val = w * grid_f;
    }
    val = warpReduceSum(val);     // Each warp performs partial reduction
    if (lane==0) shared[wid]=val; // Write reduced value to shared memory
     __syncthreads();              // Wait for all partial reductions
    //read from shared memory only if that warp existed
    val = (threadIdx.x < blockDim.x / WARP_SIZE) ? shared[lane] : 0.0;
    if (wid==0){
        val = warpReduceSum(val); // Final reduce within first warp
        if (tid == 0 && pixel_idx < n_pixel && fea_idx < d_fea){
            reconstruct_buffer_bxnxdx4[pixel_idx * d_fea * split_size + fea_idx * split_size + split_idx] = val;
        }
    }
    if (grididx < n_grid && fea_idx == (d_fea - 1) && pixel_idx < n_pixel){
        w = buffer_bxnxk[bidx][pixel_idx][grididx];
        difference = 0.0;
        for (int d = 0; d < d_fea; d++){
            grid_f = grid_fea_bxkxd[bidx][grididx][d];
            pixel_f = img_fea_bxnxd[bidx][pixel_idx][d];
            diff = line_variance_parallel_cuda_square(grid_f - pixel_f);
            difference = difference + diff;
        }
        buffer_bxnxk[bidx][pixel_idx][grididx] = w * difference; // this should be changed at the last step, other wise
    }
}

template <typename scalar_t>
__global__ void line_variance_parallel_cuda_forward_kernel_batch_sum_reduce_last_step_reconstruct(
		scalar_t* __restrict__ buffer_bxnxdx4,
		scalar_t* __restrict__ buffer_bxnxd,
		int bnum, int n_pixel, int n_grid, int d_fea, int split_size)
{
   // bidx * height + heiidx
	int presentthread = blockIdx.x * blockDim.x + threadIdx.x;
	int fea_idx = presentthread % d_fea;
	int pixel_idx = (presentthread - fea_idx) / d_fea;
	int bidx = 0;

	if (bidx >= bnum || pixel_idx >= n_pixel || fea_idx >= d_fea) {
		return;
	}
    int base_idx = bidx * n_pixel * d_fea * split_size + pixel_idx * d_fea * split_size + fea_idx * split_size;
    scalar_t sum_v = buffer_bxnxdx4[base_idx + 0];
    for (int t=1; t < split_size; t++){
        sum_v += buffer_bxnxdx4[base_idx + t];
    }
    buffer_bxnxd[bidx * n_pixel * d_fea + pixel_idx * d_fea + fea_idx] = sum_v;
}


void line_variance_parallel_cuda_forward_batch(at::Tensor img_fea_bxnxd, at::Tensor grid_fea_bxkxd, at::Tensor grid_bxkx3x2, at::Tensor img_pos_bxnx2,
                        at::Tensor variance_bxn, float sigma, at::Tensor reconstruct_bxnxd, at::Tensor buffer_bxnxk,
                        at::Tensor buffer_bxn, at::Tensor buffer_bxnx4, at::Tensor buffer_bxnxdx4, int split_size){

	int bnum = grid_bxkx3x2.size(0);
	int n_grid = grid_bxkx3x2.size(1);
	int n_pixel = img_pos_bxnx2.size(1);
	int d_fea = img_fea_bxnxd.size(2);

//    struct timeval t1, t2;
//    gettimeofday(&t1, 0);
	// for fxbxhxw image size
	// calculate the initi buffer
	const int threadnum = BLOCK_SIZE;
	const int totalthread_1 = bnum * n_pixel * n_grid;
	const int blocknum_1 = totalthread_1 / threadnum + 1;
	const dim3 threads(threadnum, 1, 1);
	const dim3 blocks_1(blocknum_1, 1, 1);
	AT_DISPATCH_FLOATING_TYPES(grid_bxkx3x2.type(), "line_variance_parallel_cuda_forward_batch_calc_buffer", ([&] {
		line_variance_parallel_cuda_forward_kernel_batch_calc_buffer<scalar_t><<<blocks_1, threads>>>(
		        img_fea_bxnxd.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
		        grid_fea_bxkxd.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
				grid_bxkx3x2.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>(),
				img_pos_bxnx2.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
                variance_bxn.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
                reconstruct_bxnxd.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
                buffer_bxnxk.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
				bnum, n_pixel, n_grid, d_fea, sigma);
	}));

    // find the maximum value in the buffer
    const int totalthread_3 = bnum * n_pixel * BLOCK_SIZE * split_size;
	const int blocknum_3 = totalthread_3 / threadnum + 1;
	const dim3 blocks_3(blocknum_3, 1, 1);
	AT_DISPATCH_FLOATING_TYPES(grid_bxkx3x2.type(), "line_variance_parallel_cuda_forward_batch_final", ([&] {
		blockReduceMax<scalar_t><<<blocks_3, threads>>>(
                buffer_bxnxk.data<scalar_t>(),
                buffer_bxnx4.data<scalar_t>(),
				bnum, n_pixel, n_grid, split_size);
	}));

	const int totalthread_4 = bnum * n_pixel;
	const int blocknum_4 = totalthread_4 / threadnum + 1;
	const dim3 blocks_4(blocknum_4, 1, 1);
	AT_DISPATCH_FLOATING_TYPES(grid_bxkx3x2.type(), "line_variance_parallel_cuda_forward_batch_final", ([&] {
		line_variance_parallel_cuda_forward_kernel_batch_max_reduce_last_step<scalar_t><<<blocks_4, threads>>>(
                buffer_bxnx4.data<scalar_t>(),
                buffer_bxn.data<scalar_t>(),
				bnum, n_pixel, n_grid, split_size);
	}));

    // find the sum of the buffer
    const int totalthread_5 = bnum * n_pixel * BLOCK_SIZE * split_size;
	const int blocknum_5 = totalthread_5 / threadnum + 1;
	const dim3 blocks_5(blocknum_5, 1, 1);
	AT_DISPATCH_FLOATING_TYPES(grid_bxkx3x2.type(), "line_variance_parallel_cuda_forward_batch_final", ([&] {
		blockReduceSum<scalar_t><<<blocks_5, threads>>>(
                buffer_bxnxk.data<scalar_t>(),
                buffer_bxnx4.data<scalar_t>(),
                buffer_bxn.data<scalar_t>(),
				bnum, n_pixel, n_grid, split_size);
	}));

    const int totalthread_6 = bnum * n_pixel;
	const int blocknum_6 = totalthread_6 / threadnum + 1;
	const dim3 blocks_6(blocknum_6, 1, 1);
	AT_DISPATCH_FLOATING_TYPES(grid_bxkx3x2.type(), "line_variance_parallel_cuda_forward_batch_final", ([&] {
		line_variance_parallel_cuda_forward_kernel_batch_sum_reduce_last_step<scalar_t><<<blocks_6, threads>>>(
                buffer_bxnx4.data<scalar_t>(),
                buffer_bxn.data<scalar_t>(),
				bnum, n_pixel, n_grid, split_size);
	}));

    const int totalthread_7 = bnum * n_pixel * BLOCK_SIZE * split_size;
    const int blocknum_7 = totalthread_7 / threadnum + 1;
    const dim3 blocks_7(blocknum_7, 1, 1);
    for (int fea_idx = 0; fea_idx < d_fea; fea_idx++){
        AT_DISPATCH_FLOATING_TYPES(grid_bxkx3x2.type(), "line_variance_parallel_cuda_forward_batch_final", ([&] {
            line_variance_parallel_cuda_forward_kernel_batch_final_1<scalar_t><<<blocks_7, threads>>>(
                    img_fea_bxnxd.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
                    grid_fea_bxkxd.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
                    grid_bxkx3x2.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>(),
                    img_pos_bxnx2.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
                    variance_bxn.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
                    buffer_bxnxk.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
                    buffer_bxn.data<scalar_t>(),
                    buffer_bxnxdx4.data<scalar_t>(),
                    bnum, n_pixel, n_grid, d_fea, fea_idx, split_size);
        }));
    }

	const int totalthread_8 = bnum * n_pixel * d_fea;
	const int blocknum_8 = totalthread_8 / threadnum + 1;
	const dim3 blocks_8(blocknum_8, 1, 1);
	AT_DISPATCH_FLOATING_TYPES(grid_bxkx3x2.type(), "line_variance_parallel_cuda_forward_batch_final", ([&] {
		line_variance_parallel_cuda_forward_kernel_batch_sum_reduce_last_step_reconstruct<scalar_t><<<blocks_8, threads>>>(
                buffer_bxnxdx4.data<scalar_t>(),
                reconstruct_bxnxd.data<scalar_t>(),
				bnum, n_pixel, n_grid, d_fea, split_size);
	}));
}