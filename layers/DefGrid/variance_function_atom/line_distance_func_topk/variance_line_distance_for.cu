#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <THC/THC.h>
#include <vector>
#include <torch/torch.h>
#include <torch/extension.h>

#define eps 1e-10
#define SCALE 1.0
#define MAX_DIS  9999999999.0
//extern THCState * state;

template<typename scalar_t>
__host__ __device__ scalar_t line_variance_topk_cuda_abs(scalar_t a){
	if (a > 0.0){
		return a;
	}
	else{
		return -a;
	}
}

template<typename scalar_t>
__host__ __device__ scalar_t line_variance_topk_cuda_square(scalar_t a){
	return a * a;
}

template<typename scalar_t>
__host__ __device__ scalar_t line_variance_topk_cuda_divide_non_zero(scalar_t a){
	if (a == 0){
		return eps;
	}
	if (a < 0){
		return a - eps;
	}
	if (a > 0){
		return a + eps;
	}
	return eps;
}

template<typename scalar_t>
__host__ __device__ scalar_t line_variance_topk_cuda_min_dis(scalar_t a, scalar_t b, scalar_t c){
	scalar_t min_d = a;
	if (b < min_d){
		min_d = b;
	}
	if (c < min_d){
		min_d = c;
	}
	return min_d;
}

template<typename scalar_t>
__host__ __device__ scalar_t line_variance_topk_cuda_min_dis_idx(scalar_t a, scalar_t b, scalar_t c){
	scalar_t min_d = a;
	int min_idx = 0;
	if (b < min_d){
		min_d = b;
		min_idx = 1;
	}
	if (c < min_d){
		min_d = c;
		min_idx = 2;
	}
	return min_idx;
}

template <typename scalar_t>
__host__ __device__ scalar_t distance_line(scalar_t x1, scalar_t y1, scalar_t x2, scalar_t y2, scalar_t x, scalar_t y){
	
	scalar_t dx1x2 = -x1 + x2;
	scalar_t dy1y2 = -y1 + y2;
	scalar_t dx1x = x - x1;
	scalar_t dy1y = y - y1;
	
	scalar_t c1 = - x * x1 + x * x2 + x1 * x1 - x1 * x2 - y * y1 + y * y2 + y1 * y1 - y1 * y2;
	scalar_t c2 = x1 * x1 - 2 * x1 * x2 + x2 * x2 + y1 * y1  - 2 * y1 * y2 + y2 * y2;
	
	scalar_t d1 = -dx1x + dx1x2 * c1 / line_variance_topk_cuda_divide_non_zero(c2);
	scalar_t d2 = -dy1y + dy1y2 * c1 / line_variance_topk_cuda_divide_non_zero(c2);
	
	scalar_t dis = 	line_variance_topk_cuda_abs(d1)	+ line_variance_topk_cuda_abs(d2);

	return dis;
}
template <typename scalar_t>
__host__ __device__ scalar_t distance_point(scalar_t x1, scalar_t y1, scalar_t x, scalar_t y){
	return line_variance_topk_cuda_abs(x - x1) + line_variance_topk_cuda_abs(y - y1);
}

template <typename scalar_t>
__host__ __device__ void distance(scalar_t* ret, scalar_t x1, scalar_t y1, scalar_t x2, scalar_t y2, scalar_t x3, scalar_t y3, scalar_t x, scalar_t y)
{
	//https://en.wikipedia.org/wiki/Barycentric_coordinate_system
	
	scalar_t x1_x2 = x1 - x2;
	scalar_t y1_y2 = y1 - y2;
	scalar_t x1_x3 = x1 - x3;
	scalar_t y1_y3 = y1 - y3;
	scalar_t x2_x3 = x2 - x3;
	scalar_t y2_y3 = y2 - y3;
	
	scalar_t x_x1 = x - x1;
	scalar_t y_y1 = y - y1;
	scalar_t x_x2 = x - x2;
	scalar_t y_y2 = y - y2;
	scalar_t x_x3 = x - x3;
	scalar_t y_y3 = y - y3;

	scalar_t k1 = y2_y3 * x_x3 - x2_x3 * y_y3;
	scalar_t k2 = x1_x3 * y_y3 - y1_y3 * x_x3;
	scalar_t k3 = y2_y3 * x1_x3 - x2_x3 * y1_y3;
	
	if(k3 == 0){ // not a legal triangle
		ret[0] = -2;
		return;
	}
	if(k3 > 0){ // clock-wise triangle
		ret[0] = -1;
		return;
	}

	//scalar_t l1 = k1 / line_variance_topk_cuda_divide_non_zero(k3);
	//scalar_t l2 = k2 / line_variance_topk_cuda_divide_non_zero(k3);
	scalar_t l1 = k1 / k3;
	scalar_t l2 = k2 / k3;
	scalar_t l3 = 1 - l1 - l2;

	scalar_t dis12 = distance_line(x1, y1, x2, y2, x, y);
	scalar_t dis23 = distance_line(x2, y2, x3, y3, x, y);
	scalar_t dis13 = distance_line(x1, y1, x3, y3, x, y);
	
	if (l1 >= 0 && l2 >= 0 && l3 >= 0){ // lie inside or on the boundary
		scalar_t min_dis_line = line_variance_topk_cuda_min_dis(dis12, dis23, dis13);
		scalar_t min_dis_line_idx = line_variance_topk_cuda_min_dis_idx(dis12, dis23, dis13);
		ret[0] = 0;
		ret[1] = min_dis_line;
		ret[2] = min_dis_line_idx;
		return;
	}

	// whether point can calculate distance to certain line
	bool within12 = ((y1_y2 * y_y1 + x_x1 * x1_x2) * (y1_y2 * y_y2 + x_x2 * x1_x2)) <= 0;
	bool within23 = ((y2_y3 * y_y3 + x_x3 * x2_x3) * (y2_y3 * y_y2 + x_x2 * x2_x3)) <= 0;	
	bool within13 = ((y1_y3 * y_y1 + x_x1 * x1_x3) * (y1_y3 * y_y3 + x_x3 * x1_x3)) <= 0;

	dis12 = within12 ? dis12 : MAX_DIS;
	dis23 = within23 ? dis23 : MAX_DIS;
	dis13 = within13 ? dis13 : MAX_DIS;

	scalar_t min_dis_line = line_variance_topk_cuda_min_dis(dis12, dis23, dis13);
	scalar_t min_dis_line_idx = line_variance_topk_cuda_min_dis_idx(dis12, dis23, dis13);
	
	scalar_t d1 = distance_point(x1, y1, x, y);
	scalar_t d2 = distance_point(x2, y2, x, y);  
	scalar_t d3 = distance_point(x3, y3, x, y);
	
	scalar_t min_dis_point = line_variance_topk_cuda_min_dis(d1, d2, d3);
	scalar_t min_dis_point_idx = line_variance_topk_cuda_min_dis_idx(d1, d2, d3);

	if (min_dis_line < min_dis_point){
		ret[0] = 1;
		ret[1] = min_dis_line;
		ret[2] = min_dis_line_idx;
	}
	else{
		ret[0] = 2;
		ret[1] = min_dis_point;
		ret[2] = min_dis_point_idx;
	}

}

template <typename scalar_t> 
__global__ void line_variance_topk_cuda_forward_kernel_batch(
        const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> img_fea_bxnxd,
        const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> grid_fea_bxkxd,
		const torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> grid_bxkx3x2,
		const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> img_pos_bxnx2,
		torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> variance_bxn,
		torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> reconstruct_bxnxd,
		torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> topk_grid_bxnxk,
		torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> buffer_bxnxk,
		int bnum, int n_pixel, int topk, int d_fea, float sigma)
{
	// bidx * height + heiidx
	int presentthread = blockIdx.x * blockDim.x + threadIdx.x;
	int pixel_idx = presentthread % n_pixel;
	int bidx = (presentthread - pixel_idx) / n_pixel;


	if (bidx >= bnum || pixel_idx >= n_pixel) {
		return;
	}

	/////////////////////////////////////////////////////////////////
	// which pixel it belongs to
	int total_idx = bidx * n_pixel * topk + pixel_idx * topk;

	scalar_t pixel_x = img_pos_bxnx2[bidx][pixel_idx][0];
	scalar_t pixel_y = img_pos_bxnx2[bidx][pixel_idx][1];

	scalar_t x0 = pixel_x * SCALE;
	scalar_t y0 = pixel_y * SCALE;
	scalar_t min_distance = 0.0;
	scalar_t find_sign = 0.0;
	scalar_t sum_exp = 0.0;
    scalar_t max_dist = -MAX_DIS;
    scalar_t ax, ay, bx, by, cx, cy;
	scalar_t condition;
	int img_pos_total_idx = bidx * n_pixel * 2 + pixel_idx * 2;
	scalar_t ret[3] = {0};
	int grididx;
	for (int k = 0; k < topk; k++){
	    grididx = __float2int_rn(topk_grid_bxnxk[bidx][pixel_idx][k]);

		ax = grid_bxkx3x2[bidx][grididx][0][0] * SCALE;
		ay = grid_bxkx3x2[bidx][grididx][0][1] * SCALE;
		bx = grid_bxkx3x2[bidx][grididx][1][0] * SCALE;
		by = grid_bxkx3x2[bidx][grididx][1][1] * SCALE;
		cx = grid_bxkx3x2[bidx][grididx][2][0] * SCALE;
		cy = grid_bxkx3x2[bidx][grididx][2][1] * SCALE;

		distance(ret, ax, ay, bx, by, cx, cy, x0, y0);
		condition = ret[0];
		min_distance = ret[1];

		if (condition < 0) {
			min_distance = - MAX_DIS;
		}
		else if (condition == 0 && find_sign == 0){
			min_distance = min_distance / sigma;
			find_sign == 1;
		}
		else{
			min_distance = - min_distance / sigma;
		}

		max_dist = max_dist > min_distance ? max_dist : min_distance;
		buffer_bxnxk[bidx][pixel_idx][k] = min_distance;
	}

	for (int k = 0; k < topk; k++){
	    buffer_bxnxk[bidx][pixel_idx][k] = expf(buffer_bxnxk[bidx][pixel_idx][k] - max_dist);
		sum_exp += buffer_bxnxk[bidx][pixel_idx][k];
	}

	scalar_t variance = 0.0;
	scalar_t grid_f = 0.0;
	scalar_t pixel_f = 0.0;
	scalar_t diff = 0.0;
	scalar_t w = 0.0;
	scalar_t difference = 0.0;
	for (int k = 0; k < topk; k++){
	    grididx = __float2int_rn(topk_grid_bxnxk[bidx][pixel_idx][k]);
		int in_sign = 0;
		if(buffer_bxnxk[bidx][pixel_idx][k] == 1){
			in_sign = 1;
		}
        buffer_bxnxk[bidx][pixel_idx][k] = buffer_bxnxk[bidx][pixel_idx][k] / (sum_exp + 1e-15);
		w = buffer_bxnxk[bidx][pixel_idx][k];
		difference = 0.0;
		for (int d = 0; d < d_fea; d++){
			grid_f = grid_fea_bxkxd[bidx][grididx][d];
			pixel_f = img_fea_bxnxd[bidx][pixel_idx][d];
			reconstruct_bxnxd[bidx][pixel_idx][d] += w * grid_f;
			diff = line_variance_topk_cuda_square(grid_f - pixel_f);
			difference = difference + diff;
		}
		variance = variance + w * difference;
		if(in_sign == 1){ //hard variance for upsample
			buffer_bxnxk[bidx][pixel_idx][k] = difference;
		}
		else{
			buffer_bxnxk[bidx][pixel_idx][k] = 0;
		}
	}
	variance_bxn[bidx][pixel_idx] = variance;
}

void line_variance_topk_cuda_forward_batch(at::Tensor img_fea_bxnxd, at::Tensor grid_fea_bxkxd, at::Tensor grid_bxkx3x2, at::Tensor img_pos_bxnx2,
                        at::Tensor variance_bxn, float sigma, at::Tensor reconstruct_bxnxd, at::Tensor topk_grid_bxnxk, at::Tensor buffer_bxnxk){

	int bnum = grid_bxkx3x2.size(0);
	int n_pixel = img_pos_bxnx2.size(1);
	int d_fea = img_fea_bxnxd.size(2);
	int topk = topk_grid_bxnxk.size(2);

	// for fxbxhxw image size
	const int threadnum = 512;
	const int totalthread = bnum * n_pixel;
	const int blocknum = totalthread / threadnum + 1;

	const dim3 threads(threadnum, 1, 1);
	const dim3 blocks(blocknum, 1, 1);


	AT_DISPATCH_FLOATING_TYPES(grid_bxkx3x2.type(), "line_variance_topk_cuda_forward_batch", ([&] {
		line_variance_topk_cuda_forward_kernel_batch<scalar_t><<<blocks, threads>>>(
		        img_fea_bxnxd.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
		        grid_fea_bxkxd.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
				grid_bxkx3x2.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>(),
				img_pos_bxnx2.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
                variance_bxn.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
                reconstruct_bxnxd.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
                topk_grid_bxnxk.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
                buffer_bxnxk.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
				bnum, n_pixel, topk, d_fea, sigma);
	}));

}
