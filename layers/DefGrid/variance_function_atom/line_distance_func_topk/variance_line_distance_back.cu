#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <THC/THC.h>
#include <vector>
#include <torch/torch.h>
#include <torch/extension.h>

#define eps 1e-10
#define SCALE 1.0
#define MAX_DIS  9999999999.0

template<typename scalar_t>
__host__ __device__ scalar_t line_variance_topk_cuda_abs(scalar_t a){
	if (a > 0.0){
		return a;
	}
	else{
		return -a;

	}
}

template<typename scalar_t>
__host__ __device__ scalar_t line_variance_topk_cuda_sign(scalar_t a){
	if (a > 0.0){
		return 1;
	}
	else if (a == 0.0){
		return 0;
	}
	else{
		return -1;
	}
}

template<typename scalar_t>
__host__ __device__ scalar_t line_variance_topk_cuda_square(scalar_t a){
	return a * a;
}


template<typename scalar_t>
__host__ __device__ scalar_t line_variance_topk_cuda_min_dis(scalar_t a, scalar_t b, scalar_t c){
	scalar_t min_d = a;
	if (b < min_d){
		min_d = b;
	}
	if (c < min_d){
		min_d = c;
	}
	return min_d;
}


template<typename scalar_t>
__host__ __device__ scalar_t line_variance_topk_cuda_min_dis_idx(scalar_t a, scalar_t b, scalar_t c){
	scalar_t min_d = a;
	int min_idx = 0;
	if (b < min_d){
		min_d = b;
		min_idx = 1;
	}
	if (c < min_d){
		min_d = c;
		min_idx = 2;
	}
	return min_idx;
}

template<typename scalar_t>
__host__ __device__ scalar_t line_variance_topk_cuda_divide_non_zero(scalar_t a){
	if (a == 0){
		return eps;
	}
	if (a < 0){
		return a - eps;
	}
	if (a > 0){
		return a + eps;
	}
	return eps;
}

template <typename scalar_t>
__host__ __device__ scalar_t distance_line(scalar_t x1, scalar_t y1, scalar_t x2, scalar_t y2, scalar_t x, scalar_t y){
	
	scalar_t dx1x2 = -x1 + x2;
	scalar_t dy1y2 = -y1 + y2;
	scalar_t dx1x = x - x1;
	scalar_t dy1y = y - y1;
	
	scalar_t c1 = - x * x1 + x * x2 + x1 * x1 - x1 * x2 - y * y1 + y * y2 + y1 * y1 - y1 * y2;
	scalar_t c2 = x1 * x1 - 2 * x1 * x2 + x2 * x2 + y1 * y1  - 2 * y1 * y2 + y2 * y2;
	
	scalar_t d1 = -dx1x + dx1x2 * c1 / line_variance_topk_cuda_divide_non_zero(c2);
	scalar_t d2 = -dy1y + dy1y2 * c1 / line_variance_topk_cuda_divide_non_zero(c2);
	
	scalar_t dis = 	line_variance_topk_cuda_abs(d1)	+ line_variance_topk_cuda_abs(d2);

	return dis;
}
template <typename scalar_t>
__host__ __device__ scalar_t distance_point(scalar_t x1, scalar_t y1, scalar_t x, scalar_t y){
	return line_variance_topk_cuda_abs(x - x1) + line_variance_topk_cuda_abs(y - y1);
}

template <typename scalar_t>
__host__ __device__ void cal_line_gradient(scalar_t* grad, scalar_t x1, scalar_t y1, scalar_t x2, scalar_t y2, scalar_t x, scalar_t y){
	
	scalar_t dx1x2 = -x1 + x2;
	scalar_t dy1y2 = -y1 + y2;
	scalar_t dx1x = x - x1;
	scalar_t dy1y = y - y1;
	
	scalar_t c1 = - x * x1 + x * x2 + x1 * x1 - x1 * x2 - y * y1 + y * y2 + y1 * y1 - y1 * y2;
	scalar_t c2 = x1 * x1 - 2 * x1 * x2 + x2 * x2 + y1 * y1  - 2 * y1 * y2 + y2 * y2;
	scalar_t c12 = c1 / line_variance_topk_cuda_divide_non_zero(c2 * c2);
	
	scalar_t cx = - dx1x - dx1x2;
	scalar_t cy = - dy1y - dy1y2;
	
	scalar_t d1 = - dx1x + dx1x2 * c1 / line_variance_topk_cuda_divide_non_zero(c2);
	scalar_t d2 = - dy1y + dy1y2 * c1 / line_variance_topk_cuda_divide_non_zero(c2);
	
	
	//scalar_t dis = line_variance_topk_cuda_abs(d1) + line_variance_topk_cuda_abs(d2);
	
	scalar_t dif_x1 = (2 * dx1x2 * dy1y2 * c12 + dy1y2 * cx / line_variance_topk_cuda_divide_non_zero(c2)) * line_variance_topk_cuda_sign(d2) + (2 * dx1x2 * dx1x2 * c12 + dx1x2 * cx / line_variance_topk_cuda_divide_non_zero(c2) + 1 - c1 / line_variance_topk_cuda_divide_non_zero(c2)) * line_variance_topk_cuda_sign(d1);
	scalar_t dif_y1 = (2 * dx1x2 * dy1y2 * c12 + dx1x2 * cy / line_variance_topk_cuda_divide_non_zero(c2)) * line_variance_topk_cuda_sign(d1) + (2 * dy1y2 * dy1y2 * c12 + dy1y2 * cy / line_variance_topk_cuda_divide_non_zero(c2) + 1 - c1 / line_variance_topk_cuda_divide_non_zero(c2)) * line_variance_topk_cuda_sign(d2);
	scalar_t dif_x2 = (dx1x * dy1y2 / line_variance_topk_cuda_divide_non_zero(c2) - 2 * dx1x2 * dy1y2 * c12) * line_variance_topk_cuda_sign(d2) + (dx1x * dx1x2 / line_variance_topk_cuda_divide_non_zero(c2) - 2 * dx1x2 * dx1x2 * c12 + c1 / line_variance_topk_cuda_divide_non_zero(c2)) * line_variance_topk_cuda_sign(d1);
	scalar_t dif_y2 = (dx1x2 * dy1y / line_variance_topk_cuda_divide_non_zero(c2) - 2 * dx1x2 * dy1y2 * c12) * line_variance_topk_cuda_sign(d1) + (dy1y * dy1y2 / line_variance_topk_cuda_divide_non_zero(c2) - 2 * dy1y2 * dy1y2 * c12 + c1 / line_variance_topk_cuda_divide_non_zero(c2)) * line_variance_topk_cuda_sign(d2);

	grad[0] = dif_x1;
	grad[1] = dif_y1;
	grad[2] = dif_x2;
	grad[3] = dif_y2;
}

template <typename scalar_t>
__host__ __device__ void distance(scalar_t* ret, scalar_t x1, scalar_t y1, scalar_t x2, scalar_t y2, scalar_t x3, scalar_t y3, scalar_t x, scalar_t y)
{
	//https://en.wikipedia.org/wiki/Barycentric_coordinate_system
	
	scalar_t x1_x2 = x1 - x2;
	scalar_t y1_y2 = y1 - y2;
	scalar_t x1_x3 = x1 - x3;
	scalar_t y1_y3 = y1 - y3;
	scalar_t x2_x3 = x2 - x3;
	scalar_t y2_y3 = y2 - y3;
	
	scalar_t x_x1 = x - x1;
	scalar_t y_y1 = y - y1;
	scalar_t x_x2 = x - x2;
	scalar_t y_y2 = y - y2;
	scalar_t x_x3 = x - x3;
	scalar_t y_y3 = y - y3;

	scalar_t k1 = y2_y3 * x_x3 - x2_x3 * y_y3;
	scalar_t k2 = x1_x3 * y_y3 - y1_y3 * x_x3;
	scalar_t k3 = y2_y3 * x1_x3 - x2_x3 * y1_y3;

	if(k3 == 0){ // not a legal triangle
		ret[0] = -2;
		return;
	}
	if(k3 > 0){ // clock-wise triangle
		ret[0] = -1;
		return;
	}

	scalar_t l1 = k1 / k3;
	scalar_t l2 = k2 / k3;
	scalar_t l3 = 1 - l1 - l2;

	scalar_t dis12 = distance_line(x1, y1, x2, y2, x, y);
	scalar_t dis23 = distance_line(x2, y2, x3, y3, x, y);
	scalar_t dis13 = distance_line(x1, y1, x3, y3, x, y);

	if (l1 >= 0 && l2 >= 0 && l3 >= 0){ // lie inside or on the boundary
		
		ret[0] = 0;
		scalar_t min_dis_line = line_variance_topk_cuda_min_dis(dis12, dis23, dis13);
		scalar_t min_dis_line_idx = line_variance_topk_cuda_min_dis_idx(dis12, dis23, dis13);
		ret[1] = min_dis_line;
		ret[2] = min_dis_line_idx;
		return;
	}
	// whether point can calculate distance to certain line
	bool within12 = ((y1_y2 * y_y1 + x_x1 * x1_x2) * (y1_y2 * y_y2 + x_x2 * x1_x2)) <= 0;
	bool within23 = ((y2_y3 * y_y3 + x_x3 * x2_x3) * (y2_y3 * y_y2 + x_x2 * x2_x3)) <= 0;	
	bool within13 = ((y1_y3 * y_y1 + x_x1 * x1_x3) * (y1_y3 * y_y3 + x_x3 * x1_x3)) <= 0;

	dis12 = within12 ? dis12 : MAX_DIS;
	dis23 = within23 ? dis23 : MAX_DIS;
	dis13 = within13 ? dis13 : MAX_DIS;

	scalar_t min_dis_line = line_variance_topk_cuda_min_dis(dis12, dis23, dis13);
	scalar_t min_dis_line_idx = line_variance_topk_cuda_min_dis_idx(dis12, dis23, dis13);

	scalar_t d1 = distance_point(x1, y1, x, y);
	scalar_t d2 = distance_point(x2, y2, x, y);  
	scalar_t d3 = distance_point(x3, y3, x, y);
	
	scalar_t min_dis_point = line_variance_topk_cuda_min_dis(d1, d2, d3);
	scalar_t min_dis_point_idx = line_variance_topk_cuda_min_dis_idx(d1, d2, d3);

	if (min_dis_line < min_dis_point){ //distance to line
		ret[0] = 1;
		ret[1] = min_dis_line;
		ret[2] = min_dis_line_idx;
	}
	else{ //distance to point
		ret[0] = 2;
		ret[1] = min_dis_point;
		ret[2] = min_dis_point_idx;
	}

}
template<typename scalar_t>
__global__ void line_variance_topk_cuda_backword_kernel_batch(
        const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> dldvariance_bxn,
		const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> img_fea_bxnxd,
        const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> grid_fea_bxkxd,
		const torch::PackedTensorAccessor<scalar_t, 4, torch::RestrictPtrTraits, size_t> grid_bxkx3x2,
		const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> img_pos_bxnx2,
		const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> dldreconstruct_bxnxd,
		torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> topk_grid_bxnxk,
		torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> buffer_bxnxk,
		scalar_t* __restrict__ dldgrid_bxkx3x2,
		int bnum, int n_pixel, int n_grid, int d_fea, float sigma, int topk){

	// bidx * height + heiidx
	int presentthread = blockIdx.x * blockDim.x + threadIdx.x;

	int pixel_idx = presentthread % n_pixel;
	int bidx = (presentthread - pixel_idx) / n_pixel;

	if (bidx >= bnum || pixel_idx >= n_pixel)
		return;

	scalar_t pixel_x = img_pos_bxnx2[bidx][pixel_idx][0];
	scalar_t pixel_y = img_pos_bxnx2[bidx][pixel_idx][1];
	scalar_t x0 = pixel_x * SCALE;
	scalar_t y0 = pixel_y * SCALE;
	scalar_t x1, y1, x2, y2; // tmp variable for calculating the gradients
	scalar_t min_distance = 0.0;
	scalar_t sum_exp = 0.0;
	int min_distance_idx = 0;
	int idx_one = 0;
	int idx_two = 0;
	scalar_t find_sign = 0.0;
	scalar_t max_dist = -MAX_DIS;
    scalar_t ax, ay, bx, by, cx, cy;

	scalar_t ret[3] = {0};
	scalar_t grad[4] = {0};
	scalar_t condition;
	int grididx = 0;
	for (int k = 0; k < topk; k++){

        grididx = __float2int_rn(topk_grid_bxnxk[bidx][pixel_idx][k]);
		ax = grid_bxkx3x2[bidx][grididx][0][0] * SCALE;
		ay = grid_bxkx3x2[bidx][grididx][0][1] * SCALE;
		bx = grid_bxkx3x2[bidx][grididx][1][0] * SCALE;
		by = grid_bxkx3x2[bidx][grididx][1][1] * SCALE;
		cx = grid_bxkx3x2[bidx][grididx][2][0] * SCALE;
		cy = grid_bxkx3x2[bidx][grididx][2][1] * SCALE;

		distance(ret, ax, ay, bx, by, cx, cy, x0, y0);
		condition = ret[0];
		min_distance = ret[1];

		if (condition < 0) {
			min_distance = - MAX_DIS;
		}
		else if (condition == 0 && find_sign ==0){
			min_distance = min_distance / sigma;
			find_sign == 1;
		}
		else{
			min_distance = - min_distance / sigma;
		}
		max_dist = max_dist > min_distance ? max_dist : min_distance;
		buffer_bxnxk[bidx][pixel_idx][k] = min_distance;
	}

	for (int k = 0; k < topk; k++){
	    buffer_bxnxk[bidx][pixel_idx][k] = expf(buffer_bxnxk[bidx][pixel_idx][k] - max_dist);
		sum_exp += buffer_bxnxk[bidx][pixel_idx][k];
	}

	scalar_t sum_gradient = 0.0;
	scalar_t pixel_f = 0.0;
	scalar_t grid_f = 0.0;
	scalar_t diff = 0.0;
	scalar_t difference = 0.0;
	scalar_t grid_f_sum = 0.0;

	for (int k = 0; k < topk; k ++){
	    buffer_bxnxk[bidx][pixel_idx][k] = buffer_bxnxk[bidx][pixel_idx][k] / line_variance_topk_cuda_divide_non_zero(sum_exp);
	    grididx = __float2int_rn(topk_grid_bxnxk[bidx][pixel_idx][k]);
	    difference = 0.0;
	    grid_f_sum = 0.0;
		for (int d = 0; d < d_fea; d++){
			grid_f = grid_fea_bxkxd[bidx][grididx][d];
			pixel_f = img_fea_bxnxd[bidx][pixel_idx][d];
			diff = line_variance_topk_cuda_square(grid_f - pixel_f);
			difference = difference + diff;
			grid_f_sum += (dldreconstruct_bxnxd[bidx][pixel_idx][d] * grid_f);
		}
	    sum_gradient += (buffer_bxnxk[bidx][pixel_idx][k] * (dldvariance_bxn[bidx][pixel_idx] * difference + \
	                                                            grid_f_sum));
	}

    find_sign = 0.0;
    scalar_t dl_dmindist_element = 0.0;
	for (int k = 0; k < topk; k++){
	    grididx = __float2int_rn(topk_grid_bxnxk[bidx][pixel_idx][k]);
        scalar_t difference = 0.0;
        scalar_t grid_f_sum = 0.0;
		for (int d = 0; d < d_fea; d++){
			grid_f = grid_fea_bxkxd[bidx][grididx][d];
			pixel_f = img_fea_bxnxd[bidx][pixel_idx][d];
			diff = line_variance_topk_cuda_square(grid_f - pixel_f);
			difference = difference + diff;
			grid_f_sum += (dldreconstruct_bxnxd[bidx][pixel_idx][d] * grid_f);
		}
        dl_dmindist_element = buffer_bxnxk[bidx][pixel_idx][k] * (dldvariance_bxn[bidx][pixel_idx] * difference + grid_f_sum) - \
                                sum_gradient * buffer_bxnxk[bidx][pixel_idx][k];

		ax = grid_bxkx3x2[bidx][grididx][0][0] * SCALE;
		ay = grid_bxkx3x2[bidx][grididx][0][1] * SCALE;
		bx = grid_bxkx3x2[bidx][grididx][1][0] * SCALE;
		by = grid_bxkx3x2[bidx][grididx][1][1] * SCALE;
		cx = grid_bxkx3x2[bidx][grididx][2][0] * SCALE;
		cy = grid_bxkx3x2[bidx][grididx][2][1] * SCALE;

		distance(ret, ax, ay, bx, by, cx, cy, x0, y0);
		condition = ret[0];
		min_distance = ret[1];
		min_distance_idx = ret[2];

		int mem_gradient_idx = bidx * n_grid * 3 * 2  + grididx * 3 * 2;
		float in_out_sign;

		if (condition < 0){
			continue;
		}
		if (condition == 0 || condition == 1){
			in_out_sign = 1 - condition * 2;
			idx_one = min_distance_idx;
			idx_two = (min_distance_idx + 1 ) % 3;
			x1 = grid_bxkx3x2[bidx][grididx][idx_one][0];
			y1 = grid_bxkx3x2[bidx][grididx][idx_one][1];
			x2 = grid_bxkx3x2[bidx][grididx][idx_two][0];
			y2 = grid_bxkx3x2[bidx][grididx][idx_two][1];
			cal_line_gradient(grad, x1, y1, x2, y2, x0, y0);
			atomicAdd((float *)(dldgrid_bxkx3x2 + (mem_gradient_idx + idx_one * 2)), float(dl_dmindist_element * grad[0] / sigma * in_out_sign));
			atomicAdd((float *)(dldgrid_bxkx3x2 + (mem_gradient_idx + idx_one * 2 + 1)), float(dl_dmindist_element * grad[1] / sigma * in_out_sign));
			atomicAdd((float *)(dldgrid_bxkx3x2 + (mem_gradient_idx + idx_two * 2)), float(dl_dmindist_element * grad[2] / sigma * in_out_sign));
			atomicAdd((float *)(dldgrid_bxkx3x2 + (mem_gradient_idx + idx_two * 2 + 1)), float(dl_dmindist_element * grad[3] / sigma * in_out_sign));
		}
		else{
			in_out_sign = -1;
			x1 = grid_bxkx3x2[bidx][grididx][min_distance_idx][0];
			y1 = grid_bxkx3x2[bidx][grididx][min_distance_idx][0];
			float signx, signy;
			if (x1 > x0){
				signx = 1;
			}
			else{
				signx = -1;
			}
			if (y1 > y0){
				signy = 1;
			}
			else{
				signy = -1;
			}
			atomicAdd((float *)(dldgrid_bxkx3x2 + (mem_gradient_idx + min_distance_idx * 2)), float(signx * dl_dmindist_element / sigma * in_out_sign));
			atomicAdd((float *)(dldgrid_bxkx3x2 + (mem_gradient_idx + min_distance_idx * 2 + 1)), float(signy * dl_dmindist_element / sigma * in_out_sign));
		}
		
	}
}

void line_variance_topk_cuda_backward_batch(at::Tensor dldvariance_bxn, at::Tensor img_fea_bxnxd, at::Tensor grid_fea_bxkxd, at::Tensor grid_bxkx3x2, at::Tensor img_pos_bxnx2,
                        float sigma, at::Tensor dldreconstruct_bxnxd, at::Tensor topk_grid_bxnxk, at::Tensor buffer_bxnxk, at::Tensor dldgrid_bxkx3x2) {

	int bnum = grid_bxkx3x2.size(0);
	int n_grid = grid_bxkx3x2.size(1);
	int n_pixel = img_pos_bxnx2.size(1);
	int d_fea = img_fea_bxnxd.size(2);
	int topk = topk_grid_bxnxk.size(2);
	// for fxbxhxw image size
	const int threadnum = 512;
	const int totalthread = bnum * n_pixel;
	const int blocknum = totalthread / threadnum + 1;

	const dim3 threads(threadnum, 1, 1);
	const dim3 blocks(blocknum, 1, 1);

	// we exchange block and thread!
	AT_DISPATCH_FLOATING_TYPES(grid_bxkx3x2.type(), "line_variance_topk_cuda_backward_batch",
			([&] {
				line_variance_topk_cuda_backword_kernel_batch<scalar_t><<<blocks, threads>>>(
				        dldvariance_bxn.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
						img_fea_bxnxd.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
                        grid_fea_bxkxd.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
                        grid_bxkx3x2.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>(),
                        img_pos_bxnx2.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
                        dldreconstruct_bxnxd.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
                        topk_grid_bxnxk.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
                        buffer_bxnxk.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
                        dldgrid_bxkx3x2.data<scalar_t>(),
                        bnum, n_pixel, n_grid, d_fea, sigma, topk);
			}));

	return;
}

